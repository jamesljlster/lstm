
#include <hip/hip_runtime.h>

#include <debug.h>

#include "lstm_cuda.h"
#include "lstm_private_cuda.h"

int lstm_clone_to_cuda(lstm_cuda_t* lstmCudaPtr, lstm_t lstm)
{
	int i, j;
	int vecLen, nodeCount;
	int ret = LSTM_NO_ERROR;

	lstm_cuda_t tmpLstmCuda = NULL;
	struct LSTM_CULAYER* dstLayerRef;
	struct LSTM_LAYER* srcLayerRef;
	struct LSTM_CONFIG_STRUCT* cfgRef;

	LOG("enter");

	// Get reference
	cfgRef = &lstm->config;

	// Create lstm cuda
	lstm_run(lstm_create_cuda(&tmpLstmCuda, cfgRef), ret, RET);

#define __lstm_clone_base_to_cuda(cuMat, base) \
	{ \
		int procIndex = cuMat * (nodeCount * vecLen) + j * vecLen; \
		if(hipMemcpy(&dstLayerRef[i].baseMat.weight[procIndex], \
					srcLayerRef[i].nodeList[j].base.weight, \
					srcLayerRef[i - 1].nodeCount * sizeof(float), \
					hipMemcpyHostToDevice) \
				!= hipSuccess) \
		{ \
			ret = LSTM_MEM_FAILED; \
			goto ERR; \
		} \
		else \
		{ \
			procIndex += srcLayerRef[i - 1].nodeCount; \
		} \
 \
		if(i == 1) \
		{ \
			if(hipMemcpy(&dstLayerRef[i].baseMat.weight[procIndex], \
						srcLayerRef[i].nodeList[j].base.rWeight, \
						srcLayerRef[cfgRef->layers - 2].nodeCount * sizeof(float), \
						hipMemcpyHostToDevice) \
					!= hipSuccess) \
			{ \
				ret = LSTM_MEM_FAILED; \
				goto ERR; \
			} \
			else \
			{ \
				procIndex += srcLayerRef[cfgRef->layers - 2].nodeCount; \
			} \
		} \
 \
		if(hipMemcpy(&dstLayerRef[i].baseMat.weight[procIndex], \
					&srcLayerRef[i].nodeList[j].base.th, \
					sizeof(float), \
					hipMemcpyHostToDevice) \
				!= hipSuccess) \
		{ \
			ret = LSTM_MEM_FAILED; \
			goto ERR; \
		} \
	}

	// Clone weight
	dstLayerRef = tmpLstmCuda->layerList;
	srcLayerRef = lstm->layerList;
	for(i = 1; i < cfgRef->layers; i++)
	{
		// Get node count and vector length
		nodeCount = srcLayerRef[i].nodeCount;
		vecLen = dstLayerRef[i].vecLen;

		// Clone node to cuda
		for(j = 0; j < nodeCount; j++)
		{
			// Clone gate network
			if(i < cfgRef->layers - 1)
			{
				__lstm_clone_base_to_cuda(LSTM_CUMAT_OG, ogNet);
				__lstm_clone_base_to_cuda(LSTM_CUMAT_FG, fgNet);
				__lstm_clone_base_to_cuda(LSTM_CUMAT_IG, igNet);
			}

			// Clone input network
			__lstm_clone_base_to_cuda(LSTM_CUMAT_INPUT, inputNet);
		}
	}

	// Assign value
	*lstmCudaPtr = tmpLstmCuda;

	goto RET;

ERR:
	lstm_delete_cuda(tmpLstmCuda);

RET:
	LOG("exit");
	return ret;
}

int lstm_clone_from_cuda(lstm_t* lstmPtr, lstm_cuda_t lstmCuda)
{
	int i, j;
	int vecLen, nodeCount;
	int ret = LSTM_NO_ERROR;

	lstm_t tmpLstm = NULL;
	struct LSTM_CULAYER* srcLayerRef;
	struct LSTM_LAYER* dstLayerRef;
	struct LSTM_CONFIG_STRUCT* cfgRef;

	LOG("enter");

	// Get reference
	cfgRef = &lstmCuda->config;

	// Create lstm
	lstm_run(lstm_create(&tmpLstm, cfgRef), ret, RET);

#define __lstm_clone_cuda_to_base(base, cuMat) \
	{ \
		int procIndex = cuMat * (nodeCount * vecLen) + j * vecLen; \
		if(hipMemcpy(dstLayerRef[i].nodeList[j].base.weight, \
					&srcLayerRef[i].baseMat.weight[procIndex], \
					dstLayerRef[i - 1].nodeCount * sizeof(float), \
					hipMemcpyDeviceToHost) \
				!= hipSuccess) \
		{ \
			ret = LSTM_MEM_FAILED; \
			goto ERR; \
		} \
		else \
		{ \
			procIndex += dstLayerRef[i - 1].nodeCount; \
		} \
 \
		if(i == 1) \
		{ \
			if(hipMemcpy(dstLayerRef[i].nodeList[j].base.rWeight, \
						&srcLayerRef[i].baseMat.weight[procIndex], \
						dstLayerRef[cfgRef->layers - 2].nodeCount * sizeof(float), \
						hipMemcpyDeviceToHost) \
					!= hipSuccess) \
			{ \
				ret = LSTM_MEM_FAILED; \
				goto ERR; \
			} \
			else \
			{ \
				procIndex += dstLayerRef[cfgRef->layers - 2].nodeCount; \
			} \
		} \
 \
		if(hipMemcpy(&dstLayerRef[i].nodeList[j].base.th, \
					&srcLayerRef[i].baseMat.weight[procIndex], \
					sizeof(float), \
					hipMemcpyDeviceToHost) \
				!= hipSuccess) \
		{ \
			ret = LSTM_MEM_FAILED; \
			goto ERR; \
		} \
	}

	// Clone weight
	srcLayerRef = lstmCuda->layerList;
	dstLayerRef = tmpLstm->layerList;
	for(i = 1; i < cfgRef->layers; i++)
	{
		// Get node count and vector length
		nodeCount = dstLayerRef[i].nodeCount;
		vecLen = srcLayerRef[i].vecLen;

		// Clone cuda node to host
		for(j = 0; j < nodeCount; j++)
		{
			// Clone gate network
			if(i < cfgRef->layers - 1)
			{
				__lstm_clone_cuda_to_base(ogNet, LSTM_CUMAT_OG);
				__lstm_clone_cuda_to_base(fgNet, LSTM_CUMAT_FG);
				__lstm_clone_cuda_to_base(igNet, LSTM_CUMAT_IG);
			}

			// Clone input network
			__lstm_clone_cuda_to_base(inputNet, LSTM_CUMAT_INPUT);
		}
	}

	// Assign value
	*lstmPtr = tmpLstm;

	goto RET;

ERR:
	lstm_delete(tmpLstm);

RET:
	LOG("exit");
	return ret;
}
