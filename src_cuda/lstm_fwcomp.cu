#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>

#include <debug.h>
#include "lstm_private_cuda.h"
#include "lstm_builtin_math_cuda.h"

//#define DUMP_CUDA
#include "lstm_dump_cuda.h"

void lstm_forward_computation_erase_cuda(lstm_cuda_t lstmCuda)
{
	int i, indexTmp;
	struct LSTM_CULAYER* layerRef;

	LOG("enter");

	// Set referenct
	layerRef = lstmCuda->layerList;

	// Set indexTmp to last hidden layer index
	indexTmp = lstmCuda->config.layers - 2;

	// Clear hidden layer outputs
	hipMemset(layerRef[indexTmp].output, 0, layerRef[indexTmp].nodeCount * sizeof(float));

	// Clear cell value
	for(i = 1; i <= indexTmp; i++)
	{
		hipMemset(layerRef[i].cell, 0, layerRef[i].nodeCount * sizeof(float));
	}

	LOG("exit");
}

// lstm_matmul<<<LSTM_CUMAT_AMOUNT * layerRef[i].nodeCount, layerRef[i].vecLen - 1>>>
__global__ void lstm_matmul(float* calcBuf, float* input, float* weight, int vecLen)
{
	calcBuf[blockIdx.x * vecLen + threadIdx.x] =
		weight[blockIdx.x * vecLen + threadIdx.x] * input[threadIdx.x];
}

// lstm_base_calc<<<LSTM_CUMAT_AMOUNT, layerRef[i].nodeCount>>>
__global__ void lstm_base_calc(float* calc, float* out, float* calcBuf, float* weight, int vecLen, int inputTFunc, int gateTFunc)
{
	int i;
	int tFuncIndex;

	// Copy threshold value to calcBuf
	calcBuf[blockIdx.x * (blockDim.x * vecLen) + (threadIdx.x + 1) * vecLen - 1] =
		weight[blockIdx.x * (blockDim.x * vecLen) + (threadIdx.x + 1) * vecLen - 1];

	// Vector reduction
	calc[blockIdx.x * blockDim.x + threadIdx.x] = 0;
	for(i = 0; i < vecLen; i++)
	{
		calc[blockIdx.x * blockDim.x + threadIdx.x] +=
			calcBuf[blockIdx.x * (blockDim.x * vecLen) + threadIdx.x * vecLen + i];
	}

	// Find network base output
	out[blockIdx.x * blockDim.x + threadIdx.x] = 0;
	tFuncIndex = (blockIdx.x == LSTM_CUMAT_INPUT) ? inputTFunc : gateTFunc;
	lstm_transfer_list_cu[tFuncIndex](&out[blockIdx.x * blockDim.x + threadIdx.x],
			calc[blockIdx.x * blockDim.x + threadIdx.x]);
}

// lstm_cell_calc<<<1, layerRef[i].nodeCount>>>
__global__ void lstm_cell_calc(float* output, float* cell, float* baseOut, int outputTFunc)
{
	float calcTmp;

	// Find cell value
	cell[threadIdx.x] = baseOut[LSTM_CUMAT_FG * blockDim.x + threadIdx.x] * cell[threadIdx.x] +
		baseOut[LSTM_CUMAT_IG * blockDim.x + threadIdx.x] *
		baseOut[LSTM_CUMAT_INPUT * blockDim.x + threadIdx.x];

	// Find output value
	lstm_transfer_list_cu[outputTFunc](&calcTmp, cell[threadIdx.x]);
	output[threadIdx.x] = baseOut[LSTM_CUMAT_OG * blockDim.x + threadIdx.x] * calcTmp;
}

void lstm_forward_computation_cuda(lstm_cuda_t lstmCuda, float* input, float* output)
{
	int i;
	int indexTmp;

	struct LSTM_CONFIG_STRUCT* cfgRef;
	struct LSTM_CULAYER* layerRef;

	LOG("enter");

	// Set referenct
	cfgRef = &lstmCuda->config;
	layerRef = lstmCuda->layerList;

	// Copy inputs
	hipMemcpy(layerRef[0].output, input,
			cfgRef->inputs * sizeof(float),
			hipMemcpyHostToDevice);

	// Copy recurrent output
	indexTmp = cfgRef->layers - 2;
	hipMemcpy(&layerRef[0].output[cfgRef->inputs], layerRef[indexTmp].output,
			layerRef[indexTmp].nodeCount * sizeof(float),
			hipMemcpyDeviceToDevice);

	DUMP("layerRef[0].output: ", layerRef[0].output, layerRef[0].nodeCount);

	// Hidden layer calculation
	for(i = 1; i < cfgRef->layers - 1; i++)
	{
		// Matrix element multiplication
		lstm_matmul<<<LSTM_CUMAT_AMOUNT * layerRef[i].nodeCount, layerRef[i].vecLen - 1>>>(
				layerRef[i].baseMat.calcBuf,
				layerRef[i - 1].output,
				layerRef[i].baseMat.weight,
				layerRef[i].vecLen);
		//hipDeviceSynchronize();

		lstm_base_calc<<<LSTM_CUMAT_AMOUNT, layerRef[i].nodeCount>>>(
				layerRef[i].baseMat.calc,
				layerRef[i].baseMat.out,
				layerRef[i].baseMat.calcBuf,
				layerRef[i].baseMat.weight,
				layerRef[i].vecLen,
				layerRef[i].inputTFunc,
				layerRef[i].gateTFunc);
		//hipDeviceSynchronize();

		// LSTM cell calculation
		lstm_cell_calc<<<1, layerRef[i].nodeCount>>>(
				layerRef[i].output,
				layerRef[i].cell,
				layerRef[i].baseMat.out,
				layerRef[i].outputTFunc);
		//hipDeviceSynchronize();
	}

	// Output layer calculation
	lstm_matmul<<<layerRef[i].nodeCount, layerRef[i].vecLen - 1>>>(
			layerRef[i].baseMat.calcBuf,
			layerRef[i - 1].output,
			layerRef[i].baseMat.weight,
			layerRef[i].vecLen);
	//hipDeviceSynchronize();

	lstm_base_calc<<<1, layerRef[i].nodeCount>>>(
			layerRef[i].baseMat.calc,
			layerRef[i].baseMat.out,
			layerRef[i].baseMat.calcBuf,
			layerRef[i].baseMat.weight,
			layerRef[i].vecLen,
			layerRef[i].inputTFunc,
			layerRef[i].gateTFunc);
	//hipDeviceSynchronize();

	// Get output
	if(output != NULL)
	{
		hipMemcpy(output, layerRef[i].baseMat.out, cfgRef->outputs * sizeof(float), hipMemcpyDeviceToHost);
	}
	else
	{
		hipDeviceSynchronize();
	}

	LOG("exit");
}

