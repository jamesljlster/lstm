#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include <lstm_builtin_math_cuda.h>

int main()
{
	int i;
	float* cuPtr;
	float in, out, cuOut, err;

	hipError_t cuErr;

	hipMalloc(&cuPtr, sizeof(float));

	for(i = 0; i < LSTM_TFUNC_AMOUNT; i++)
	{
		err = 0;
		for(in = -1; in <= 1; in += 0.1)
		{
			out = lstm_transfer_list[i](in);
			run_tfunc<<<1, 1>>>(cuPtr, in, i);
			hipDeviceSynchronize();
			hipMemcpy(&cuOut, cuPtr, sizeof(float), hipMemcpyDeviceToHost);

			err += fabs(out - cuOut);
		}

		printf("%s error: %lf\n", lstm_transfer_func_name[i], err);
		cuErr = hipGetLastError();
		printf("Last cuda error: %s, %s\n", hipGetErrorName(cuErr), hipGetErrorString(cuErr));
		printf("\n");
	}

	return 0;
}
